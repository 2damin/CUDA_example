#include "hip/hip_runtime.h"



#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <time.h>
#include <sstream>
#include <iostream>

#ifndef __CUDACC_RTC__
#define __CUDACC_RTC__
#endif

#include <hip/device_functions.h>

using namespace std;

#define imin(a, b) (a<b? a:b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c) {

	__shared__ float cache[threadsPerBlock];
	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0.0;

	while (tid < N) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = temp;

	__syncthreads();

	int i = blockDim.x / 2;

	while (i != 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] = cache[cacheIndex] + cache[cacheIndex + i];
			}
		__syncthreads();
		i = i / 2;
	}

	if (cacheIndex == 0) {
		c[blockIdx.x] = cache[0];
	}
}

int main(void) {
	float *a, *b, *c, result, cpu_result;
	float *dev_a, *dev_b, *dev_c;
	

	a = new float[N];
	b = new float[N];
	c = new float[N];
	result = 0;
	cpu_result = 0;
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * 2;
		c[i] = 0;
	}
	cout << "start" << endl;
	
	for (int i = 0; i < N; i++) {
		cpu_result += a[i] * b[i];
	}

	
	cout << cpu_result << endl;

	hipMalloc((void**)&dev_a, sizeof(float)*N);
	hipMalloc((void**)&dev_b, sizeof(float)*N);
	hipMalloc((void**)&dev_c, sizeof(float)*blocksPerGrid);

	hipMemcpy(dev_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemset(dev_c, 0, sizeof(float)*blocksPerGrid);


	dot <<< blocksPerGrid, threadsPerBlock >>> (dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, sizeof(float)*blocksPerGrid, hipMemcpyDeviceToHost);

	for (int j = 0; j < blocksPerGrid; j++)
	{
		result += c[j];
	}


	std::cout << result << std::endl;

	system("pause");

	hipDeviceReset();

	delete[] a;
	delete[] b;
	delete[] c;

	return 0;
}